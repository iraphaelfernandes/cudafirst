#include "hip/hip_runtime.h"
    %%cu

    #include "hip/hip_runtime.h"
    #include ""

    #include <stdio.h>

    //matriz quadrada
    #define N 3

    // 1 0 0
    // 0 1 0
    // 0 0 1
    
    
    void printM(int L, int C, int* mat) {

        int i, j;

        for (i = 0; i < L; i++)
        {
            for (j = 0; j < C; j++)
            {
                printf("%d  ", mat[i*C + j]);
            }
            printf("\n");
        }
        printf("\n\n");
    }



    __global__ void ident(int n, int* mat) {
        
        int i = blockDim.x * blockIdx.x + threadIdx.x; //i representa a coluna de um elemento qualquer
        int j = blockDim.y * blockIdx.y + threadIdx.y; //j representa a linha de um elemento qualquer
        
        int id = j*n + i; //n é a dimensão, posto

        mat[id] = 0;

        if (i < n && j < n) {
        
            if (i == j) {
                
                mat[id] = 1;
            } 
        }
    }



    int main() {

        int* mat = (int*)malloc(N * N * sizeof(int*));

        int* matGPU = 0;
        hipError_t cudaStatus;

        // Alocar espaço na memória do device
        cudaStatus = hipMalloc( ( void** ) &matGPU , N * N * sizeof( int ) );

        dim3 grid(2, 2);
        dim3 block(3, 3);

        ident<<<grid, block>>>(N, matGPU); //como ele executa isso?
        

        cudaStatus = hipMemcpy( mat , matGPU , N * N * sizeof( int ) , hipMemcpyDeviceToHost );
        
        printM(N, N, mat);

        return 0;
    }