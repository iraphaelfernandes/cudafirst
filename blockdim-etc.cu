#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


// int xindex = threadIdx.x + blockIdx.x * blockDim.x;
__global__ void meu_kernel( void ){
  
  printf( "ID da thread: %d\n" , threadIdx.x );
  printf( "Dim do bloco: %d\n" , blockDim.x );
  printf( "ID bloco: %d\n" , blockIdx.x);
}

// Define a variável de captura de erros
hipError_t cudaStatus;

int main( ){

// Informa o device a ser usado caso exista mais de 1
cudaStatus = hipSetDevice( 0 );

// Testa a função hipSetDevice retornou erro
if ( cudaStatus != hipSuccess ){
  
    fprintf( stderr , "hipSetDevice falhou! Existe dispositivo com suporte a CUDA instalado?" );
    goto Error;
  }


meu_kernel << <1 , 10 >> > ( );


// Captura o último erro ocorrido
cudaStatus = hipGetLastError( );
if ( cudaStatus != hipSuccess ){
  
  fprintf( stderr , "meu_kernel falhou: %s\n" ,
    hipGetErrorString( cudaStatus ) );
  goto Error;
}

// Sincroniza a execução do kernel com a CPU
cudaStatus = hipDeviceSynchronize( );
if ( cudaStatus != hipSuccess ){
  
  fprintf( stderr , "hipDeviceSynchronize retornou erro %d após lançamento do kernel!\n" ,
      cudaStatus );
  goto Error;
}

int nDevices;

  // hipGetDeviceCount(&nDevices);
  // for (int i = 0; i < nDevices; i++) {
  //   hipDeviceProp_t prop;
  //   hipGetDeviceProperties(&prop, i);
  //   printf("Device Number: %d\n", i);
  //   printf("  Device name: %s\n", prop.name);
  //   printf("  Memory Clock Rate (KHz): %d\n",
  //          prop.memoryClockRate);
  //   printf("  Memory Bus Width (bits): %d\n",
  //          prop.memoryBusWidth);
  //   printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
  //          2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  // }

Error:
// Executa a limpeza GPU
cudaStatus = hipDeviceReset ( );
if ( cudaStatus != hipSuccess )
{
  fprintf( stderr , "hipDeviceReset falhou!"  );
  return 1;
}

  return 0 ;
 
}