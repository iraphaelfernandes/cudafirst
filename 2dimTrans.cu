#include "hip/hip_runtime.h"
%%cu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//dimensões da matriz
#define LINHAS 3
#define COLUNAS 3

//dimensões da grid
#define DIMX 3
#define DIMY 3

__global__ void transp(int linha, int coluna, int* matin, int* matout) {
  
    int x = blockDim.x * blockIdx.x + threadIdx.x; //coluna
    int y = blockDim.y * blockIdx.y + threadIdx.y; //linha

    
    if(y < 2)  {
      printf("");
    }

    if(x<COLUNAS && y < LINHAS)  {
        
      int idin = y*coluna + x;
      int idout = x*linha + y;

      matout[idout] = matin[idin]; 
    }
}

void printM(int l, int c, int* mat) {

    int i, j;

    for (i = 0; i < l; i++)
    {
        for (j = 0; j < c; j++)
        {
            printf("%d\t", mat[i*c + j]); //retirei a ideia do indice daqui
        }
        printf("\n\n");
    }
    printf("\n\n");
}

int main() {
    const int nElem = LINHAS*COLUNAS; //número de elementos na matriz

    int* matin = (int*)malloc(nElem * sizeof(int*));
    int* matout = (int*)malloc(nElem * sizeof(int*));

    int elemento = 1;
    for(int i = 0; i < nElem; i++){
      
      matin[i] = elemento;
      elemento++;
    }

    printM(LINHAS, COLUNAS, matin);

    int* matinGPU = 0;
    int* matoutGPU = 0;

    hipError_t cudaStatus;
    
    cudaStatus = hipMalloc( ( void** ) &matinGPU , nElem * sizeof( int ) );
    cudaStatus = hipMalloc( ( void** ) &matoutGPU , nElem * sizeof( int ) );

    cudaStatus = hipMemcpy( matinGPU , matin , nElem * sizeof( int ) , hipMemcpyHostToDevice );

    dim3 grid(DIMX, DIMY);

    dim3 block( (COLUNAS + DIMX - 1)/DIMX , (LINHAS + DIMY - 1)/DIMY);

    transp<<<grid, block>>>(LINHAS, COLUNAS, matinGPU, matoutGPU);

    cudaStatus = hipMemcpy( matout, matoutGPU, nElem * sizeof( int ), hipMemcpyDeviceToHost);
    
    printM(COLUNAS, LINHAS, matout); //Pq não posso imprimir o que está na gpu?

    return 0;
}