#include "hip/hip_runtime.h"
%%cu

#include <stdio.h>
#include <stdlib.h>

//dimensões da matriz
#define L 4
#define C 5

//dimensões da grid
#define BLOCKDIMX 3
#define BLOCKDIMY 2

//1 dimensão pq não consigo passar 2 dimensoes do host para o device
// int matrizEntrada[];
// int matrizSaida[];

void printMatriz(int l, int c, int* mat) {
    
    int i, j;

    for (i = 0; i < l; i++){
        
        for (j = 0; j < c; j++){
            
            printf("%d\t", mat[i*c + j]);
        }
        printf("\n\n");
    }
    printf("\n\n");
}

__global__ void invertePosicaoDasLinhas(int* matrizEntrada, int* matrizSaida) {
    
    int elementoSaida, elementoEntrada;
    
    //coluna de um elemento qualquer de matin
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    //linha de um elemento qualquer de matin
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    
    elementoSaida = i*C+j;
    elementoEntrada = (i+1)*C+j;
    matrizSaida[elementoSaida]=matrizEntrada[elementoEntrada];


    if(i==(L-1)) {
        
        elementoEntrada=j;
        elementoSaida=i*C+j;
        matrizSaida[elementoSaida]=matrizEntrada[elementoEntrada];
    }
}

int main() {

    float time;
    hipEvent_t start, stop;
    
    const int len = L*C; //número de elementos na matriz
    
    //1 dimensão pq não consigo passar 2 dimensoes do host para o device
    int* matin = (int*)malloc(len * sizeof(int*));
    int* matout = (int*)malloc(len * sizeof(int*));
    
    hipError_t cudaStatus;

    //Preenche a matriz de entrada aleatoriamente
    for(int i = 0; i < len; i++){
        matin[i] = (rand() % 99) + 1;
    }
    
    //antes da função principal, depois das declarações de matrizes e vars
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    
    
    
    int* dev_matin = 0;
    int* dev_matout = 0;
    // Alocar espaço na memória do device
    cudaStatus = hipMalloc( ( void** ) &dev_matin , len * sizeof( int ) );
    cudaStatus = hipMalloc( ( void** ) &dev_matout , len * sizeof( int ) );

    // Copia matin para a memória do device
    cudaStatus = hipMemcpy( dev_matin , matin , len * sizeof( int ) , hipMemcpyHostToDevice );

    dim3 block(BLOCKDIMX, BLOCKDIMY);
    dim3 grid( (C + BLOCKDIMX - 1)/BLOCKDIMX , (L + BLOCKDIMY - 1)/BLOCKDIMY);

    
    invertePosicaoDasLinhas<<<grid, block>>>(dev_matin, dev_matout);


    //depois da função principal
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    //Copia matout para a memória do host
    cudaStatus = hipMemcpy( matout, dev_matout, len * sizeof( int ), hipMemcpyDeviceToHost);


    
    
    //----------------Imprimir----------------------
    printf("Matriz de entrada:\n\n");
    printMatriz(L, C, matin);
    
    printf("Matriz de saída:\n\n");
    printMatriz(L, C, matout);
    
    //depois de tudo
    printf("Time to generate:  %3.5f ms \n", time);
    
}