#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"



int main() {
  int nDevices;
  nDevices = 5;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Nome do divice: %s\n", prop.name);
    printf("  Taxa de clock da memória (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Largura do barramento de memória (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Pico de largura de banda de memória (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
